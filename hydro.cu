
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define GAMMA (5.0 / 3.0)
#define min2(a, b) (a) < (b) ? (a) : (b)
#define max2(a, b) (a) > (b) ? (a) : (b)

// Convert primitive quantities (density, velocity, and pressure) to conserved
// quantities (density, momentum, and energy).
void primitive_to_conserved(double *prim, double *cons) {
  double density  = prim[0];
  double velocity = prim[1];
  double pressure = prim[2];

  double momentum = velocity * density;
  double energy   = (0.5 * density * velocity * velocity) +
                    (pressure / (GAMMA - 1));

  cons[0] = density;
  cons[1] = momentum;
  cons[2] = energy;
}

// Convert conserved quantities (density, momentum, and energy) to primitive
// quantities (density, velocity, and pressure).
void conserved_to_primitive(double *cons, double *prim) {
  double density  = cons[0];
  double momentum = cons[1];
  double energy   = cons[2];

  double velocity = momentum / density;
  double pressure = (energy - (0.5 * density * velocity * velocity)) *
                    (GAMMA - 1);

  prim[0] = density;
  prim[1] = velocity;
  prim[2] = pressure;
}

// Compute the sound speed of the medium.
double cs(double *prim) {
  double density  = prim[0];
  double pressure = prim[2];

  return sqrt(GAMMA*pressure/density);
}

// Compute outer wave speeds.
void speed(double *ul, double *ur, double *s) {
	double pl[3];
	double pr[3];

	conserved_to_primitive(ul, pl);
	conserved_to_primitive(ur, pr);

	double vl = pl[1];
	double vr = pr[1];

	s[0] = min2((vl - cs(pl)), (vr - cs(pr)));
  s[1] = max2((vl + cs(pl)), (vr + cs(pr)));
}

// Compute wave speed in star region.
double speed_star(double *ul, double *ur, double sl, double sr) {
  double pl[3];
	double pr[3];

	conserved_to_primitive(ul, pl);
	conserved_to_primitive(ur, pr);

	double rhol = pl[0];
	double vl   = pl[1];
	double prel = pl[2];
	double rhor = pr[0];
	double vr   = pr[1];
	double prer = pr[2];

	double num = prer - prel + rhol * vl * (sl - vl) - rhor * vr * (sr - vr);
	double den = rhol * (sl - vl) - rhor * (sr - vr);

	return num / den;
}

// Compute cell flux vector.
void flux_vector(double *cons, double *flux) {
	double prim[3];
	conserved_to_primitive(cons, prim);

  double rho = prim[0];
	double v = prim[1];
	double pre = prim[2];

	flux[0] = rho * v;
	flux[1] = rho * v * v + pre;
	flux[2] = v * (pre + 0.5 * rho * v * v + pre / (GAMMA - 1));
}

// Compute flux vector in star region.
void flux_star_vector(double *cons, double *d_star, double *flux, double s_star,
                      double s, double *flux_star) {
  double prim[3];
	conserved_to_primitive(cons, prim);

  double rho = prim[0];
	double v   = prim[1];
	double pre = prim[2];

  for (int i = 0; i < 3; ++i) {
    flux_star[i] = (s_star * (s * cons[i] - flux[i]) + s * (pre + rho*(s - v) *
                   (s_star - v)) * d_star[i]) / (s - s_star);
  }
}

// Compute HLLC interface flux.
void hllc_flux(double *ul, double *ur, double *flux_half){
  double s[2];
  speed(ul, ur, s);
	double s_l       = s[0];
  double s_r       = s[1];
	double s_star    = speed_star(ul, ur, s_l, s_r);
	double d_star[3] = {0, 1, s_star};
  double fl[3];
  double fr[3];
  double fl_star[3];
  double fr_star[3];

	flux_vector(ul, fl);
	flux_vector(ur, fr);
	flux_star_vector(ul, d_star, fl, s_star, s_l, fl_star);
	flux_star_vector(ur, d_star, fr, s_star, s_r, fr_star);

	if (0 <= s_l) {
	  flux_half[0] = fl[0];
    flux_half[1] = fl[1];
    flux_half[2] = fl[2];
  } else if ((s_l <= 0) && (0 < s_star)) {
	  flux_half[0] = fl_star[0];
    flux_half[1] = fl_star[1];
    flux_half[2] = fl_star[2];
  } else if ((s_star <= 0) && (0 < s_r)) {
	  flux_half[0] = fr_star[0];
    flux_half[1] = fr_star[1];
    flux_half[2] = fr_star[2];
  } else {
	  flux_half[0] = fr[0];
    flux_half[1] = fr[1];
    flux_half[2] = fr[2];
  }
}

// Establish initial conditions for primitive quantities.
void initialize_primitive(double *primitive, double dx, int n, double x0) {
  for (int i = 0; i < n; ++i) {
    double x = x0 + (i + 0.5) * dx;
    double *prim = &primitive[3*i];
    if (x < 0.5) {
      prim[0] = 1;
      prim[1] = 0;
      prim[2] = 1;
    } else {
      prim[0] = 0.1;
      prim[1] = 0;
      prim[2] = 0.125;
    }
  }
}

int main() {
  const double tmax  = 0.5;
  const int n        = 1000;
  const double xl    = 0;
  const double xr    = 1;
  const double dx    = (xr - xl) / n;
  const double dt    = 0.00025;
  const double chkpt = 0.0025;

  double primitive[3*n];
  double conserved[3*n];
  double conserved1[3*n];

  initialize_primitive(primitive, dx, n, xl);
  for (int i = 0; i < n; ++i) {
    double *prim = &primitive[3*i];
    double *cons = &conserved[3*i];
    primitive_to_conserved(prim, cons);
  }

  double t = 0;
  int j = 0;

  // Evolve the simulation in time.
  while (t < tmax) {
    // Update the simulation in space.
    for (int i = 1; i < (n-1); ++i) {
			double *cons_im1 = &conserved[3*(i-1)];
    	double *cons_i   = &conserved[3*(i)];
    	double *cons_ip1 = &conserved[3*(i+1)];

      double f_iph[3];
      double f_imh[3];
			hllc_flux(cons_im1, cons_i, f_imh);
      hllc_flux(cons_i, cons_ip1, f_iph);

			conserved1[3*i]   = conserved[3*i] - (f_iph[0] - f_imh[0]) * dt / dx;
      conserved1[3*i+1] = conserved[3*i+1] - (f_iph[1] - f_imh[1]) * dt / dx;
      conserved1[3*i+2] = conserved[3*i+2] - (f_iph[2] - f_imh[2]) * dt / dx;
    }

    for (int i = 3; i < (3*(n-1)); ++i) {
			conserved[i] = conserved1[i];
    }

    // Save conserved vectors to text files in checkpoint intervals.
    if (t >= (chkpt * j)) {
      FILE *fp;
      char filepath[256];
      snprintf (filepath, sizeof(filepath), "output/data%d.txt", j);
      fp = fopen(filepath, "w");
      for (int k = 0; k < n; ++k) {
        fprintf(fp, "%f %f %f %f\n", (k + 0.5) * dx, conserved[3*k],
                conserved[3*k+1], conserved[3*k+2]);
      }
      fclose(fp);
      j += 1;
    } else {
      ;
    }

    t += dt;
  }

  return 0;
}
