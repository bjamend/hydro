#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define GAMMA (5.0 / 3.0)
#define PI 3.14159
#define min2(a, b) (a) < (b) ? (a) : (b)
#define max2(a, b) (a) > (b) ? (a) : (b)

// Convert primitive quantities (density, velocity, and pressure) to conserved
// quantities (density, momentum, and energy).
void primitive_to_conserved(double *prim, double *cons) {
  double density  = prim[0];
  double velocity = prim[1];
  double pressure = prim[2];

  double momentum = velocity * density;
  double energy   = (0.5 * density * velocity * velocity) +
                    (pressure / (GAMMA - 1));

  cons[0] = density;
  cons[1] = momentum;
  cons[2] = energy;
}

// Convert conserved quantities (density, momentum, and energy) to primitive
// quantities (density, velocity, and pressure).
void conserved_to_primitive(double *cons, double *prim) {
  double density  = cons[0];
  double momentum = cons[1];
  double energy   = cons[2];

  double velocity = momentum / density;
  double pressure = (energy - (0.5 * density * velocity * velocity)) *
                    (GAMMA - 1);

  prim[0] = density;
  prim[1] = velocity;
  prim[2] = pressure;
}

// Compute the sound speed of the medium.
double cs(double *prim) {
  double density  = prim[0];
  double pressure = prim[2];

  return sqrt(GAMMA*pressure/density);
}

// Compute outer wave speeds.
void speed(double *ul, double *ur, double *s) {
	double pl[3];
	double pr[3];

	conserved_to_primitive(ul, pl);
	conserved_to_primitive(ur, pr);

	double vl = pl[1];
	double vr = pr[1];

	s[0] = min2((vl - cs(pl)), (vr - cs(pr)));
  s[1] = max2((vl + cs(pl)), (vr + cs(pr)));
}

// Compute wave speed in star region.
double speed_star(double *ul, double *ur, double sl, double sr) {
  double pl[3];
	double pr[3];

	conserved_to_primitive(ul, pl);
	conserved_to_primitive(ur, pr);

	double rhol = pl[0];
	double vl   = pl[1];
	double prel = pl[2];
	double rhor = pr[0];
	double vr   = pr[1];
	double prer = pr[2];

	double num = prer - prel + rhol * vl * (sl - vl) - rhor * vr * (sr - vr);
	double den = rhol * (sl - vl) - rhor * (sr - vr);

	return num / den;
}

// Compute cell flux vector.
void flux_vector(double *cons, double *flux) {
	double prim[3];
	conserved_to_primitive(cons, prim);

  double rho = prim[0];
	double v = prim[1];
	double pre = prim[2];

	flux[0] = rho * v;
	flux[1] = rho * v * v + pre;
	flux[2] = v * (pre + 0.5 * rho * v * v + pre / (GAMMA - 1));
}

// Compute flux vector in star region.
void flux_star_vector(double *cons, double *d_star, double *flux, double s_star,
                      double s, double *flux_star) {
  double prim[3];
	conserved_to_primitive(cons, prim);

  double rho = prim[0];
	double v   = prim[1];
	double pre = prim[2];

  for (int i = 0; i < 3; ++i) {
    flux_star[i] = (s_star * (s * cons[i] - flux[i]) + s * (pre + rho*(s - v) *
                   (s_star - v)) * d_star[i]) / (s - s_star);
  }
}

double sign(double a) {
  if (a < 0) {
    return -1;
  } else if (a > 0) {
    return 1;
  } else {
    return 0;
  }
}

double minmod(double a, double b, double c) {
  return abs(sign(a) + sign(b)) * (sign(a) + sign(c)) * (min2(a, (min(b, c)))) / 4;
}

// Compute HLLC interface flux.
void hllc_flux(double *ul2, double *ul1, double *ur1, double *ur2, double *flux_half, double plm_theta, double *s){
  double pl2[3];
  double pl1[3];
  double pr1[3];
  double pr2[3];
  conserved_to_primitive(ul2, pl2);
  conserved_to_primitive(ul1, pl1);
  conserved_to_primitive(ur1, pr1);
  conserved_to_primitive(ur2, pr2);
  double pl[3];
  double pr[3];
  double ul[3];
  double ur[3];

  for (int i = 0; i < 3; ++i) {
    pl[i] = pl1[i] + 0.5 * minmod(plm_theta * (pl1[i] - pl2[i]),
                                  0.5 * (pr1[i] - pl2[i]),
                                  plm_theta * (pr1[i] - pl1[i]));
    pr[i] = pr1[i] - 0.5 * minmod(plm_theta * (pr1[i] - pl1[i]),
                                  0.5 * (pr2[i] - pl1[i]),
                                  plm_theta * (pr2[i] - pr1[i]));
  }

  primitive_to_conserved(pl, ul);
  primitive_to_conserved(pr, ur);

  speed(ul, ur, s);
	double s_l       = s[0];
  double s_r       = s[1];
	double s_star    = speed_star(ul, ur, s_l, s_r);
	double d_star[3] = {0, 1, s_star};
  double fl[3];
  double fr[3];
  double fl_star[3];
  double fr_star[3];

	flux_vector(ul, fl);
	flux_vector(ur, fr);
	flux_star_vector(ul, d_star, fl, s_star, s_l, fl_star);
	flux_star_vector(ur, d_star, fr, s_star, s_r, fr_star);

	if (0 <= s_l) {
	  flux_half[0] = fl[0];
    flux_half[1] = fl[1];
    flux_half[2] = fl[2];
  } else if ((s_l <= 0) && (0 < s_star)) {
	  flux_half[0] = fl_star[0];
    flux_half[1] = fl_star[1];
    flux_half[2] = fl_star[2];
  } else if ((s_star <= 0) && (0 < s_r)) {
	  flux_half[0] = fr_star[0];
    flux_half[1] = fr_star[1];
    flux_half[2] = fr_star[2];
  } else {
	  flux_half[0] = fr[0];
    flux_half[1] = fr[1];
    flux_half[2] = fr[2];
  }
}

double gaussian(double x) {
  double mu = 0.5;
  double sigma = 0.2;
  return (1/sigma/sqrt(2*PI)) * exp(-(x-mu)*(x-mu)/2/sigma/sigma);
}

// Establish initial conditions for primitive quantities.
void initialize_primitive(double *primitive, double dx, int n, double x0) {
  for (int i = 0; i < n; ++i) {
    double x = x0 + (i + 0.5) * dx;
    double *prim = &primitive[3*i];
    prim[0] = gaussian(x);
    prim[1] = 1;
    prim[2] = 1;
  }
}

int main() {
  const double tmax  = 0.5;
  const int n        = 1000;
  const double xl    = 0;
  const double xr    = 1;
  const double dx    = (xr - xl) / n;
  const double chkpt = 0.0025;
  double plm_theta   = 1.5;
  //double cfl_number  = 0.4;

  double primitive[3*n];
  double conserved[3*n];
  double conserved1[3*n];
  double conserved2[3*n];
  double conserved3[3*n];

  initialize_primitive(primitive, dx, n, xl);
  for (int i = 0; i < n; ++i) {
    double *prim = &primitive[3*i];
    double *cons = &conserved[3*i];
    double *cons1 = &conserved1[3*i];
    double *cons2 = &conserved2[3*i];
    primitive_to_conserved(prim, cons);
    primitive_to_conserved(prim, cons1);
    primitive_to_conserved(prim, cons2);
  }

  for (int i = 0; i < 6; ++i) {
    conserved1[i] = conserved[i];
    conserved2[i] = conserved[i];
    conserved1[3*n-i-1] = conserved[3*n-i-1];
    conserved2[3*n-i-1] = conserved[3*n-i-1];
  }

  double t = 0;
  int j = 0;
  double dt = 0.00001;
  double s[2];
  //double a1 = 0;

  // Evolve the simulation in time.
  while (t < tmax) {
    //double a = 0;
    // Update the simulation in space.
    for (int i = 2; i < (n-2); ++i) {
      double *cons_im2 = &conserved[3*(i-2)];
			double *cons_im1 = &conserved[3*(i-1)];
    	double *cons_i00 = &conserved[3*(i+0)];
    	double *cons_ip1 = &conserved[3*(i+1)];
      double *cons_ip2 = &conserved[3*(i+2)];
      double f_iph[3];
      double f_imh[3];
			hllc_flux(cons_im2, cons_im1, cons_i00, cons_ip1, f_imh, plm_theta, s);
      hllc_flux(cons_im1, cons_i00, cons_ip1, cons_ip2, f_iph, plm_theta, s);

			conserved1[3*i+0] = conserved[3*i+0] - (f_iph[0] - f_imh[0]) * dt / dx;
      conserved1[3*i+1] = conserved[3*i+1] - (f_iph[1] - f_imh[1]) * dt / dx;
      conserved1[3*i+2] = conserved[3*i+2] - (f_iph[2] - f_imh[2]) * dt / dx;

      double *cons_im2_1 = &conserved1[3*(i-2)];
      double *cons_im1_1 = &conserved1[3*(i-1)];
      double *cons_i00_1 = &conserved1[3*(i+0)];
      double *cons_ip1_1 = &conserved1[3*(i+1)];
      double *cons_ip2_1 = &conserved1[3*(i+2)];
      double f_iph1[3];
      double f_imh1[3];
      hllc_flux(cons_im2_1, cons_im1_1, cons_i00_1, cons_ip1_1, f_imh1, plm_theta, s);
      hllc_flux(cons_im1_1, cons_i00_1, cons_ip1_1, cons_ip2_1, f_iph1, plm_theta, s);

      conserved2[3*i+0] = 3 * conserved[3*i+0] / 4 + conserved1[3*i+0] / 4 -
                          (f_iph1[0] - f_imh1[0]) * dt / dx / 4;
      conserved2[3*i+1] = 3 * conserved[3*i+1] / 4 + conserved1[3*i+1] / 4 -
                          (f_iph1[1] - f_imh1[1]) * dt / dx / 4;
      conserved2[3*i+2] = 3 * conserved[3*i+2] / 4 + conserved1[3*i+2] / 4 -
                          (f_iph1[2] - f_imh1[2]) * dt / dx / 4;

      double *cons_im2_2 = &conserved2[3*(i-2)];
      double *cons_im1_2 = &conserved2[3*(i-1)];
      double *cons_i00_2 = &conserved2[3*(i+0)];
      double *cons_ip1_2 = &conserved2[3*(i+1)];
      double *cons_ip2_2 = &conserved2[3*(i+2)];
      double f_iph2[3];
      double f_imh2[3];
      hllc_flux(cons_im2_2, cons_im1_2, cons_i00_2, cons_ip1_2, f_imh2, plm_theta, s);
      hllc_flux(cons_im1_2, cons_i00_2, cons_ip1_2, cons_ip2_2, f_iph2, plm_theta, s);

      conserved3[3*i+0] = conserved[3*i+0] / 3 + 2 * conserved2[3*i+0] / 3 -
                          2 * (f_iph2[0] - f_imh2[0]) * dt / dx / 3;
      conserved3[3*i+1] = conserved[3*i+1] / 3 + 2 * conserved2[3*i+1] / 3 -
                          2 * (f_iph2[1] - f_imh2[1]) * dt / dx / 3;
      conserved3[3*i+2] = conserved[3*i+2] / 3 + 2 * conserved2[3*i+2] / 3 -
                          2 * (f_iph2[2] - f_imh2[2]) * dt / dx / 3;

      //a1 = max2(abs(s[0]), abs(s[1]));
      //a = max2(a1, abs(a));
    }

    // Save conserved vectors to text files in checkpoint intervals.
    if (t >= (chkpt * j)) {
      FILE *fp;
      char filepath[256];
      snprintf (filepath, sizeof(filepath), "output/data%d.txt", j);
      fp = fopen(filepath, "w");
      for (int k = 0; k < n; ++k) {
        fprintf(fp, "%f %f %f %f\n", xl + (k + 0.5) * dx, conserved[3*k],
                conserved[3*k+1], conserved[3*k+2]);
      }
      fclose(fp);
      j += 1;
    } else {
      ;
    }

    for (int i = 6; i < (3*(n-2)); ++i) {
      conserved[i] = conserved3[i];
    }

    //double dt = cfl_number * dx / a;

    t += dt;
  }

  return 0;
}
